
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void blend_kernel(float *out, const float *in, int DIM)
{
    int N = DIM + 2; // physical row length (ghosts included)
    int pitchY = N;

    int i = 1 + blockIdx.x * blockDim.x + threadIdx.x;
    int j = 1 + blockIdx.y * blockDim.y + threadIdx.y;
    if (i > DIM || j > DIM)
        return;

    int idx = i + j * pitchY;

    // six neighbours (all legal because of ghost layer)
    float left = in[idx - 1];
    float right = in[idx + 1];
    float back = in[idx - pitchY];
    float front = in[idx + pitchY];

    out[idx] = (left + right + back + front) * (1.0f / 4.0f);
}

__global__ void copy_const_kernel(float *iptr, float *source, int DIM)
{
    int N = DIM + 2; // physical row length (ghosts included)
    int pitchY = N;

    int i = 1 + blockIdx.x * blockDim.x + threadIdx.x;
    int j = 1 + blockIdx.y * blockDim.y + threadIdx.y;
    if (i > DIM || j > DIM)
        return;

    int idx = i + j * pitchY;
    if (i == 1 || i == DIM || j == 1 || j == DIM) // edge
        iptr[idx] = source[idx];
}

int main(int argc, char **argv)
{
    if (argc != 3) {
        printf("Usage: %s <maxiter> <T>\n", argv[0]);
        return 1;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int const DIM = 1024;
    int const maxiter = atoi(argv[1]);
    int const T = atoi(argv[2]);
    int const Nside = DIM + 2;

    size_t N = Nside * Nside; // include ghost cell to avoid branch divergence
    float *h_src = (float *)malloc(N * sizeof(float));
    for (int j = 0; j < Nside; ++j)
        for (int i = 0; i < Nside; ++i) {
            int idx = i + j * Nside;
            h_src[idx] = 273.f;
        }
    // set physical top edge (row DIM) to 400 K
    for (int i = 1; i <= DIM; ++i)
        h_src[i + DIM * Nside] = 400.f;

    float *d_out, *d_in, *d_src;
    hipMalloc(&d_out, N * sizeof(float));
    hipMalloc(&d_in, N * sizeof(float));
    hipMalloc(&d_src, N * sizeof(float));

    hipMemcpy(d_in, h_src, N * sizeof(float), hipMemcpyHostToDevice);  // interior+edges
    hipMemcpy(d_out, h_src, N * sizeof(float), hipMemcpyHostToDevice); // safe first swap
    hipMemcpy(d_src, h_src, N * sizeof(float), hipMemcpyHostToDevice); // immutable boundary

    dim3 blocks((DIM + T - 1) / T, (DIM + T - 1) / T);
    dim3 threads(T, T);

    float total_time = 0;
    for (int iter = 0; iter < maxiter; ++iter) {
        // captrue the start time
        hipEventRecord(start, 0);

        copy_const_kernel<<<blocks, threads>>>(d_in, d_src, DIM);
        blend_kernel<<<blocks, threads>>>(d_out, d_in, DIM); // write "out" from "in"

        // get stop time, and display timing results
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float gpu_time;
        hipEventElapsedTime(&gpu_time, start, stop);
        total_time += gpu_time;

        float *tmp = d_out;
        d_out = d_in;
        d_in = tmp;
    }
    printf("time per update: %3.5f ms\n", total_time / maxiter);

    hipMemcpy(h_src, d_in, sizeof(float) * N, hipMemcpyDeviceToHost);

    free(h_src);
    hipFree(d_src);
    hipFree(d_in);
    hipFree(d_out);
    return 0;
}