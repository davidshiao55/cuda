#include "hip/hip_runtime.h"
#include "../common/book.h"

#define imin(a, b) (a < b ? a : b)
#define sum_squares(x) (x * (x + 1) * (2 * x + 1) / 6)

const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void dot(int size, float *a, float *b, float *c)
{
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    float tmp = 0;
    while (tid < size) {
        tmp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    // set the cache value
    cache[cacheIndex] = tmp;

    // synchronize threads in this block
    __syncthreads();

    // reduction
    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }
    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}

float malloc_test(int size)
{
    hipEvent_t start, stop;
    float *a, *b, c, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;
    float elapsedTime;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    // allocate memory on the CPU side
    a = (float *)malloc(sizeof(float) * size);
    b = (float *)malloc(sizeof(float) * size);
    partial_c = (float *)malloc(sizeof(float) * blocksPerGrid);

    // allocate the memory on the GPU
    HANDLE_ERROR(hipMalloc(&dev_a, sizeof(float) * size));
    HANDLE_ERROR(hipMalloc(&dev_b, sizeof(float) * size));
    HANDLE_ERROR(hipMalloc(&dev_partial_c, sizeof(float) * blocksPerGrid));

    // fill in the host memory with data
    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    HANDLE_ERROR(hipEventRecord(start, 0));
    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR(hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice));

    dot<<<blocksPerGrid, threadsPerBlock>>>(size, dev_a, dev_b, dev_partial_c);

    // copy the arrays 'partial_c' back to the CPU
    HANDLE_ERROR(hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

    // finish up on the CPU side
    c = 0;
    for (int i = 0; i < blocksPerGrid; i++) {
        c += partial_c[i];
    }

    // free memory on GPU
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_partial_c);

    // free memory on CPU
    free(a);
    free(b);
    free(partial_c);

    // free Event
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    printf("Value calculated: %f\n", c);

    return elapsedTime;
}

float cuda_host_alloc_test(int size)
{
    hipEvent_t start, stop;
    float *a, *b, c, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;
    float elapsedTime;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    // allocate memory on the CPU side
    HANDLE_ERROR(hipHostAlloc(&a, sizeof(float) * size, hipHostMallocWriteCombined | hipHostMallocMapped));
    HANDLE_ERROR(hipHostAlloc(&b, sizeof(float) * size, hipHostMallocWriteCombined | hipHostMallocMapped));
    HANDLE_ERROR(hipHostAlloc(&partial_c, sizeof(float) * blocksPerGrid, hipHostMallocMapped));

    // allocate the memory on the GPU
    HANDLE_ERROR(hipHostGetDevicePointer(&dev_a, a, 0));
    HANDLE_ERROR(hipHostGetDevicePointer(&dev_b, b, 0));
    HANDLE_ERROR(hipHostGetDevicePointer(&dev_partial_c, partial_c, 0));

    // fill in the host memory with data
    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    HANDLE_ERROR(hipEventRecord(start, 0));
    // copy the arrays 'a' and 'b' to the GPU

    dot<<<blocksPerGrid, threadsPerBlock>>>(size, dev_a, dev_b, dev_partial_c);
    HANDLE_ERROR(hipDeviceSynchronize());

    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

    // finish up on the CPU side
    c = 0;
    for (int i = 0; i < blocksPerGrid; i++) {
        c += partial_c[i];
    }

    HANDLE_ERROR(hipHostFree(a));
    HANDLE_ERROR(hipHostFree(b));
    HANDLE_ERROR(hipHostFree(partial_c));

    // free Event
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    printf("Value calculated: %f\n", c);

    return elapsedTime;
}

int main()
{
    hipDeviceProp_t prop;
    int whichDevice;
    HANDLE_ERROR(hipGetDevice(&whichDevice));
    HANDLE_ERROR(hipGetDeviceProperties(&prop, whichDevice));
    if (prop.canMapHostMemory != 1) {
        printf("Device cannot map memory\n");
        return 0;
    }
    HANDLE_ERROR(hipSetDeviceFlags(hipDeviceMapHost));

    float elapsedTime = malloc_test(N);
    printf("Time using hipMalloc %3.5f ms\n", elapsedTime);

    elapsedTime = cuda_host_alloc_test(N);
    printf("Time using hipHostAlloc: %3.5f ms\n", elapsedTime);
}